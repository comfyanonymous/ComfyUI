#include "hip/hip_runtime.h"
// from https://github.com/rosinality/stylegan2-pytorch/blob/master/op/fused_bias_act_kernel.cu
// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html

#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
static __global__ void fused_bias_act_kernel(scalar_t* out, const scalar_t* p_x, const scalar_t* p_b, const scalar_t* p_ref,
    int act, int grad, scalar_t alpha, scalar_t scale, int loop_x, int size_x, int step_b, int size_b, int use_bias, int use_ref) {
    int xi = blockIdx.x * loop_x * blockDim.x + threadIdx.x;

    scalar_t zero = 0.0;

    for (int loop_idx = 0; loop_idx < loop_x && xi < size_x; loop_idx++, xi += blockDim.x) {
        scalar_t x = p_x[xi];

        if (use_bias) {
            x += p_b[(xi / step_b) % size_b];
        }

        scalar_t ref = use_ref ? p_ref[xi] : zero;

        scalar_t y;

        switch (act * 10 + grad) {
            default:
            case 10: y = x; break;
            case 11: y = x; break;
            case 12: y = 0.0; break;

            case 30: y = (x > 0.0) ? x : x * alpha; break;
            case 31: y = (ref > 0.0) ? x : x * alpha; break;
            case 32: y = 0.0; break;
        }

        out[xi] = y * scale;
    }
}


torch::Tensor fused_bias_act_op(const torch::Tensor& input, const torch::Tensor& bias, const torch::Tensor& refer,
    int act, int grad, float alpha, float scale) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    auto x = input.contiguous();
    auto b = bias.contiguous();
    auto ref = refer.contiguous();

    int use_bias = b.numel() ? 1 : 0;
    int use_ref = ref.numel() ? 1 : 0;

    int size_x = x.numel();
    int size_b = b.numel();
    int step_b = 1;

    for (int i = 1 + 1; i < x.dim(); i++) {
        step_b *= x.size(i);
    }

    int loop_x = 4;
    int block_size = 4 * 32;
    int grid_size = (size_x - 1) / (loop_x * block_size) + 1;

    auto y = torch::empty_like(x);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "fused_bias_act_kernel", [&] {
        fused_bias_act_kernel<scalar_t><<<grid_size, block_size, 0, stream>>>(
            y.data_ptr<scalar_t>(),
            x.data_ptr<scalar_t>(),
            b.data_ptr<scalar_t>(),
            ref.data_ptr<scalar_t>(),
            act,
            grad,
            alpha,
            scale,
            loop_x,
            size_x,
            step_b,
            size_b,
            use_bias,
            use_ref
        );
    });

    return y;
}
