#include "hip/hip_runtime.h"
// from https://github.com/rosinality/stylegan2-pytorch/blob/master/op/upfirdn2d_kernel.cu
// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html

#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static __host__ __device__ __forceinline__ int floor_div(int a, int b) {
  int c = a / b;

  if (c * b > a) {
    c--;
  }

  return c;
}

struct UpFirDn2DKernelParams {
  int up_x;
  int up_y;
  int down_x;
  int down_y;
  int pad_x0;
  int pad_x1;
  int pad_y0;
  int pad_y1;

  int major_dim;
  int in_h;
  int in_w;
  int minor_dim;
  int kernel_h;
  int kernel_w;
  int out_h;
  int out_w;
  int loop_major;
  int loop_x;
};

template <typename scalar_t>
__global__ void upfirdn2d_kernel_large(scalar_t *out, const scalar_t *input,
                                       const scalar_t *kernel,
                                       const UpFirDn2DKernelParams p) {
  int minor_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int out_y = minor_idx / p.minor_dim;
  minor_idx -= out_y * p.minor_dim;
  int out_x_base = blockIdx.y * p.loop_x * blockDim.y + threadIdx.y;
  int major_idx_base = blockIdx.z * p.loop_major;

  if (out_x_base >= p.out_w || out_y >= p.out_h ||
      major_idx_base >= p.major_dim) {
    return;
  }

  int mid_y = out_y * p.down_y + p.up_y - 1 - p.pad_y0;
  int in_y = min(max(floor_div(mid_y, p.up_y), 0), p.in_h);
  int h = min(max(floor_div(mid_y + p.kernel_h, p.up_y), 0), p.in_h) - in_y;
  int kernel_y = mid_y + p.kernel_h - (in_y + 1) * p.up_y;

  for (int loop_major = 0, major_idx = major_idx_base;
       loop_major < p.loop_major && major_idx < p.major_dim;
       loop_major++, major_idx++) {
    for (int loop_x = 0, out_x = out_x_base;
         loop_x < p.loop_x && out_x < p.out_w; loop_x++, out_x += blockDim.y) {
      int mid_x = out_x * p.down_x + p.up_x - 1 - p.pad_x0;
      int in_x = min(max(floor_div(mid_x, p.up_x), 0), p.in_w);
      int w = min(max(floor_div(mid_x + p.kernel_w, p.up_x), 0), p.in_w) - in_x;
      int kernel_x = mid_x + p.kernel_w - (in_x + 1) * p.up_x;

      const scalar_t *x_p =
          &input[((major_idx * p.in_h + in_y) * p.in_w + in_x) * p.minor_dim +
                 minor_idx];
      const scalar_t *k_p = &kernel[kernel_y * p.kernel_w + kernel_x];
      int x_px = p.minor_dim;
      int k_px = -p.up_x;
      int x_py = p.in_w * p.minor_dim;
      int k_py = -p.up_y * p.kernel_w;

      scalar_t v = 0.0f;

      for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {
          v += static_cast<scalar_t>(*x_p) * static_cast<scalar_t>(*k_p);
          x_p += x_px;
          k_p += k_px;
        }

        x_p += x_py - w * x_px;
        k_p += k_py - w * k_px;
      }

      out[((major_idx * p.out_h + out_y) * p.out_w + out_x) * p.minor_dim +
          minor_idx] = v;
    }
  }
}

template <typename scalar_t, int up_x, int up_y, int down_x, int down_y,
          int kernel_h, int kernel_w, int tile_out_h, int tile_out_w>
__global__ void upfirdn2d_kernel(scalar_t *out, const scalar_t *input,
                                 const scalar_t *kernel,
                                 const UpFirDn2DKernelParams p) {
  const int tile_in_h = ((tile_out_h - 1) * down_y + kernel_h - 1) / up_y + 1;
  const int tile_in_w = ((tile_out_w - 1) * down_x + kernel_w - 1) / up_x + 1;

  __shared__ volatile float sk[kernel_h][kernel_w];
  __shared__ volatile float sx[tile_in_h][tile_in_w];

  int minor_idx = blockIdx.x;
  int tile_out_y = minor_idx / p.minor_dim;
  minor_idx -= tile_out_y * p.minor_dim;
  tile_out_y *= tile_out_h;
  int tile_out_x_base = blockIdx.y * p.loop_x * tile_out_w;
  int major_idx_base = blockIdx.z * p.loop_major;

  if (tile_out_x_base >= p.out_w | tile_out_y >= p.out_h |
      major_idx_base >= p.major_dim) {
    return;
  }

  for (int tap_idx = threadIdx.x; tap_idx < kernel_h * kernel_w;
       tap_idx += blockDim.x) {
    int ky = tap_idx / kernel_w;
    int kx = tap_idx - ky * kernel_w;
    scalar_t v = 0.0;

    if (kx < p.kernel_w & ky < p.kernel_h) {
      v = kernel[(p.kernel_h - 1 - ky) * p.kernel_w + (p.kernel_w - 1 - kx)];
    }

    sk[ky][kx] = v;
  }

  for (int loop_major = 0, major_idx = major_idx_base;
       loop_major < p.loop_major & major_idx < p.major_dim;
       loop_major++, major_idx++) {
    for (int loop_x = 0, tile_out_x = tile_out_x_base;
         loop_x < p.loop_x & tile_out_x < p.out_w;
         loop_x++, tile_out_x += tile_out_w) {
      int tile_mid_x = tile_out_x * down_x + up_x - 1 - p.pad_x0;
      int tile_mid_y = tile_out_y * down_y + up_y - 1 - p.pad_y0;
      int tile_in_x = floor_div(tile_mid_x, up_x);
      int tile_in_y = floor_div(tile_mid_y, up_y);

      __syncthreads();

      for (int in_idx = threadIdx.x; in_idx < tile_in_h * tile_in_w;
           in_idx += blockDim.x) {
        int rel_in_y = in_idx / tile_in_w;
        int rel_in_x = in_idx - rel_in_y * tile_in_w;
        int in_x = rel_in_x + tile_in_x;
        int in_y = rel_in_y + tile_in_y;

        scalar_t v = 0.0;

        if (in_x >= 0 & in_y >= 0 & in_x < p.in_w & in_y < p.in_h) {
          v = input[((major_idx * p.in_h + in_y) * p.in_w + in_x) *
                        p.minor_dim +
                    minor_idx];
        }

        sx[rel_in_y][rel_in_x] = v;
      }

      __syncthreads();
      for (int out_idx = threadIdx.x; out_idx < tile_out_h * tile_out_w;
           out_idx += blockDim.x) {
        int rel_out_y = out_idx / tile_out_w;
        int rel_out_x = out_idx - rel_out_y * tile_out_w;
        int out_x = rel_out_x + tile_out_x;
        int out_y = rel_out_y + tile_out_y;

        int mid_x = tile_mid_x + rel_out_x * down_x;
        int mid_y = tile_mid_y + rel_out_y * down_y;
        int in_x = floor_div(mid_x, up_x);
        int in_y = floor_div(mid_y, up_y);
        int rel_in_x = in_x - tile_in_x;
        int rel_in_y = in_y - tile_in_y;
        int kernel_x = (in_x + 1) * up_x - mid_x - 1;
        int kernel_y = (in_y + 1) * up_y - mid_y - 1;

        scalar_t v = 0.0;

#pragma unroll
        for (int y = 0; y < kernel_h / up_y; y++)
#pragma unroll
          for (int x = 0; x < kernel_w / up_x; x++)
            v += sx[rel_in_y + y][rel_in_x + x] *
                 sk[kernel_y + y * up_y][kernel_x + x * up_x];

        if (out_x < p.out_w & out_y < p.out_h) {
          out[((major_idx * p.out_h + out_y) * p.out_w + out_x) * p.minor_dim +
              minor_idx] = v;
        }
      }
    }
  }
}

torch::Tensor upfirdn2d_op(const torch::Tensor &input,
                           const torch::Tensor &kernel, int up_x, int up_y,
                           int down_x, int down_y, int pad_x0, int pad_x1,
                           int pad_y0, int pad_y1) {
  int curDevice = -1;
  hipGetDevice(&curDevice);
  hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

  UpFirDn2DKernelParams p;

  auto x = input.contiguous();
  auto k = kernel.contiguous();

  p.major_dim = x.size(0);
  p.in_h = x.size(1);
  p.in_w = x.size(2);
  p.minor_dim = x.size(3);
  p.kernel_h = k.size(0);
  p.kernel_w = k.size(1);
  p.up_x = up_x;
  p.up_y = up_y;
  p.down_x = down_x;
  p.down_y = down_y;
  p.pad_x0 = pad_x0;
  p.pad_x1 = pad_x1;
  p.pad_y0 = pad_y0;
  p.pad_y1 = pad_y1;

  p.out_h = (p.in_h * p.up_y + p.pad_y0 + p.pad_y1 - p.kernel_h + p.down_y) /
            p.down_y;
  p.out_w = (p.in_w * p.up_x + p.pad_x0 + p.pad_x1 - p.kernel_w + p.down_x) /
            p.down_x;

  auto out =
      at::empty({p.major_dim, p.out_h, p.out_w, p.minor_dim}, x.options());

  int mode = -1;

  int tile_out_h = -1;
  int tile_out_w = -1;

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 4 && p.kernel_w <= 4) {
    mode = 1;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 3 && p.kernel_w <= 3) {
    mode = 2;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 2 && p.up_y == 2 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 4 && p.kernel_w <= 4) {
    mode = 3;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 2 && p.up_y == 2 && p.down_x == 1 && p.down_y == 1 &&
      p.kernel_h <= 2 && p.kernel_w <= 2) {
    mode = 4;
    tile_out_h = 16;
    tile_out_w = 64;
  }

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 2 && p.down_y == 2 &&
      p.kernel_h <= 4 && p.kernel_w <= 4) {
    mode = 5;
    tile_out_h = 8;
    tile_out_w = 32;
  }

  if (p.up_x == 1 && p.up_y == 1 && p.down_x == 2 && p.down_y == 2 &&
      p.kernel_h <= 2 && p.kernel_w <= 2) {
    mode = 6;
    tile_out_h = 8;
    tile_out_w = 32;
  }

  dim3 block_size;
  dim3 grid_size;

  if (tile_out_h > 0 && tile_out_w > 0) {
    p.loop_major = (p.major_dim - 1) / 16384 + 1;
    p.loop_x = 1;
    block_size = dim3(32 * 8, 1, 1);
    grid_size = dim3(((p.out_h - 1) / tile_out_h + 1) * p.minor_dim,
                     (p.out_w - 1) / (p.loop_x * tile_out_w) + 1,
                     (p.major_dim - 1) / p.loop_major + 1);
  } else {
    p.loop_major = (p.major_dim - 1) / 16384 + 1;
    p.loop_x = 4;
    block_size = dim3(4, 32, 1);
    grid_size = dim3((p.out_h * p.minor_dim - 1) / block_size.x + 1,
                     (p.out_w - 1) / (p.loop_x * block_size.y) + 1,
                     (p.major_dim - 1) / p.loop_major + 1);
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "upfirdn2d_cuda", [&] {
    switch (mode) {
    case 1:
      upfirdn2d_kernel<scalar_t, 1, 1, 1, 1, 4, 4, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 2:
      upfirdn2d_kernel<scalar_t, 1, 1, 1, 1, 3, 3, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 3:
      upfirdn2d_kernel<scalar_t, 2, 2, 1, 1, 4, 4, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 4:
      upfirdn2d_kernel<scalar_t, 2, 2, 1, 1, 2, 2, 16, 64>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 5:
      upfirdn2d_kernel<scalar_t, 1, 1, 2, 2, 4, 4, 8, 32>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    case 6:
      upfirdn2d_kernel<scalar_t, 1, 1, 2, 2, 4, 4, 8, 32>
          <<<grid_size, block_size, 0, stream>>>(out.data_ptr<scalar_t>(),
                                                 x.data_ptr<scalar_t>(),
                                                 k.data_ptr<scalar_t>(), p);

      break;

    default:
      upfirdn2d_kernel_large<scalar_t><<<grid_size, block_size, 0, stream>>>(
          out.data_ptr<scalar_t>(), x.data_ptr<scalar_t>(),
          k.data_ptr<scalar_t>(), p);
    }
  });

  return out;
}
